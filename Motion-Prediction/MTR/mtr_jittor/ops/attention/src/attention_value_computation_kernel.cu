/*
Transformer function helper function.
Written by tomztyang,
2021/08/23
*/

#include <math.h>
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include "attention_func.h"

#define THREADS_PER_BLOCK 256
#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
// #define DEBUG


__global__ void attention_value_computation_forward(
    int b, int total_query_num, int local_size,
    int total_key_num, int nhead, int hdim,
    const int *query_batch_cnt, const int *key_batch_cnt, const int* index_pair_batch,
    const int *index_pair,
    const float *attn_weight, const float* value_features,
    float *output) {
    // dim3 blocks(DIVUP(total_query_num * local_size, THREADS_PER_BLOCK), nhead, hdim);
    // params query_batch_cnt: [b]
    // params key_batch_cnt: [b]
    // params index_pair_batch: [total_query_num]
    // params index_pair: [total_query_num, local_size]
    // params attn_weight: [total_query_num, local_size, nhead]
    // params value_features: [total_key_num, nhead, hdim]
    // params output: [total_query_num, nhead, hdim]

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int head_idx = blockIdx.y;
    int hdim_idx = blockIdx.z;
    if (index >= total_query_num * local_size ||
        head_idx >= nhead ||
        hdim_idx >= hdim) return;

    if (index_pair[index] == -1){
        // Ignore index.
        return;
    }

    int query_idx = index / local_size;
    int batch_idx = index_pair_batch[query_idx];
    int key_start_idx = 0;
    for (int i = 0; i < batch_idx; i++){
        key_start_idx += key_batch_cnt[i];
    }

    // 1. Obtain value features.
    key_start_idx += index_pair[index];
    value_features += key_start_idx * nhead * hdim + head_idx * hdim + hdim_idx;
    // 2. Obtain attention weight.
    attn_weight += index * nhead + head_idx;
    // 3. Do dot product.
    output += query_idx * nhead * hdim + head_idx * hdim + hdim_idx;
    atomicAdd(
        output,
        attn_weight[0] * value_features[0]);
}


void attention_value_computation_launcher(
    int b, int total_query_num, int local_size,
    int total_key_num, int nhead, int hdim,
    const int *query_batch_cnt, const int *key_batch_cnt, const int* index_pair_batch,
    const int *index_pair,
    const float *attn_weight, const float* value_features,
    float *output){
    // params query_batch_cnt: [b]
    // params key_batch_cnt: [b]
    // params index_pair_batch: [total_query_num]
    // params index_pair: [total_query_num, local_size]
    // params attn_weight: [total_query_num, local_size, nhead]
    // params value_features: [total_key_num, nhead, hdim]
    // params output: [total_query_num, nhead, hdim]

    dim3 blocks(DIVUP(total_query_num * local_size, THREADS_PER_BLOCK), nhead, hdim);
    dim3 threads(THREADS_PER_BLOCK);
    attention_value_computation_forward<<<blocks, threads>>>(
        b, total_query_num, local_size, total_key_num, nhead, hdim,
        query_batch_cnt, key_batch_cnt, index_pair_batch,
        index_pair, attn_weight, value_features,
        output);
}


__global__ void attention_value_computation_backward(
    int b, int total_query_num, int local_size,
    int total_key_num, int nhead, int hdim,
    const int *query_batch_cnt, const int *key_batch_cnt, const int* index_pair_batch,
    const int *index_pair,
    const float *attn_weight, const float* value_features,
    float *grad_out, float * grad_attn_weight, float * grad_value_features) {
    // dim3 blocks(DIVUP(total_query_num * local_size, THREADS_PER_BLOCK), nhead, hdim);
    // params query_batch_cnt: [b]
    // params key_batch_cnt: [b]
    // params index_pair_batch: [total_query_num]
    // params index_pair: [total_query_num, local_size]
    // params attn_weight: [total_query_num, local_size, nhead]
    // params value_features: [total_key_num, nhead, hdim]
    // params grad_out: [total_query_num, nhead, hdim]
    // params grad_attn_weight: [total_query_num, local_size, nhead]
    // params grad_value_features: [total_key_num, nhead, hdim]

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int head_idx = blockIdx.y;
    int hdim_idx = blockIdx.z;
    if (index >= total_query_num * local_size ||
        head_idx >= nhead ||
        hdim_idx >= hdim) return;

    if (index_pair[index] == -1){
        // Ignore index.
        return;
    }

    int query_idx = index / local_size;
    int batch_idx = index_pair_batch[query_idx];
    int key_start_idx = 0;
    for (int i = 0; i < batch_idx; i++){
        key_start_idx += key_batch_cnt[i];
    }

    // 1. Obtain value features.
    key_start_idx += index_pair[index];
    value_features += key_start_idx * nhead * hdim + head_idx * hdim + hdim_idx;
    grad_value_features += key_start_idx * nhead * hdim + head_idx * hdim + hdim_idx;
    // 2. Obtain attention weight.
    attn_weight += index * nhead + head_idx;
    grad_attn_weight += index * nhead + head_idx;

    // 3. Obtain grad out.
    grad_out += query_idx * nhead * hdim + head_idx * hdim + hdim_idx;
    atomicAdd(
        grad_attn_weight,
        grad_out[0] * value_features[0]);
    atomicAdd(
        grad_value_features,
        grad_out[0] * attn_weight[0]);
}


void attention_value_computation_grad_launcher(
    int b, int total_query_num, int local_size,
    int total_key_num, int nhead, int hdim,
    const int *query_batch_cnt, const int *key_batch_cnt, const int* index_pair_batch,
    const int *index_pair,
    const float *attn_weight, const float* value_features,
    float *grad_out, float* grad_attn_weight, float* grad_value_features){
    // params query_batch_cnt: [b]
    // params key_batch_cnt: [b]
    // params index_pair_batch: [total_query_num]
    // params index_pair: [total_query_num, local_size]
    // params attn_weight: [total_query_num, local_size, nhead]
    // params value_features: [total_key_num, nhead, hdim]
    // params grad_out: [total_query_num, nhead, hdim]
    // params grad_attn_weight: [total_query_num, local_size, nhead]
    // params grad_value_features: [total_key_num, nhead, hdim]

    dim3 blocks(DIVUP(total_query_num * local_size, THREADS_PER_BLOCK), nhead, hdim);
    dim3 threads(THREADS_PER_BLOCK);
    attention_value_computation_backward<<<blocks, threads>>>(
        b, total_query_num, local_size, total_key_num, nhead, hdim,
        query_batch_cnt, key_batch_cnt, index_pair_batch,
        index_pair, attn_weight, value_features,
        grad_out, grad_attn_weight, grad_value_features);
}